#include "hip/hip_runtime.h"
﻿
#include <cmath>
#include <memory>
#include <exception>

#include <hip/hip_runtime.h>

#include "forward_gpu.h"
#include "device_ptr.h"
#include "device_array.h"
#include "cuda_helper.h"
#include "../data/data.h"
#include "../global/global.h"

void forward_gpu::init_cuda_device()
{
	int device_count;
	auto err = hipGetDeviceCount(&device_count);
	CHECK;
	global::log("forward", "init_cuda_device complete");
}

__global__ void test_device_kernel(float_t *a, float_t *b, float_t *c, int num)
{
	int i = threadIdx.x;
	if (i >= num)
	{
		return;
	}
	c[i] = a[i] * b[i];
}

//计算正演kernel函数
//a: float_t，回线半径(m)
//i0: float_t，发射电流(A)
//h: float_t，发射、接收回线高度(m)
//cosine: device_array，余弦变换系数
//hankel: device_array，汉克尔变换系数
//resistence: device_array，地层电阻率
//height: device_array，地层厚度
//time: device_array，时间
//response: device_array，输出响应
__global__ void forward_kernel(float_t a, float_t i0, float_t h,
	device_array cosine,
	device_array hankel,
	device_array resistence,
	device_array height,
	device_array time,
	device_array reponse)
{
	constexpr float_t mu0 = global::mu0;
	constexpr float_t a1 = -7.91001919000e+00;
	constexpr float_t s1 = 8.79671439570e-02;

	int time_idx = blockIdx.x;
	int cosine_idx = threadIdx.x;

	__shared__ float_t t;
	//每个block中的第一个线程为计算准备数据
	if (cosine_idx == 0)
	{
		t = time[time_idx];
	}
	__sync_threads();
}

void forward_gpu::test_cuda_device()
{
	global::scoped_timer("test_cuda");

	device_ptr da;
	device_ptr db;
	device_ptr dc;

	float_t a[] = { 1,2,3,4,5 };
	float_t b[] = { 1,2,3,4,5 };

	auto size = sizeof(a) / sizeof(float_t);
	float_ptr c(new float_t[size]);

	da.allocate(size);
	db.allocate(size);
	dc.allocate(size);

	copy_to_device(a, da.get(), size);
	copy_to_device(b, db.get(), size);

	test_device_kernel <<<1, 32 >>> (da, db, dc, size);

	copy_to_host(dc.get(), c.get(), size);

	for (auto i = 0; i < size; ++i)
	{
		if (a[i] * b[i] != c[i])
		{
			throw std::runtime_error("测试cuda设备失败，计算错误");
		}
	}
}

forward_base::forward_data forward_gpu::forward()
{
	global::scoped_timer("forward");

	return forward_data();
}