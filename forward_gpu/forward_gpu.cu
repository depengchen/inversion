#include "hip/hip_runtime.h"
﻿#include <cmath>
#include <complex>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include "forward_gpu.h"
#include "device_array.h"
#include "cuda_helper.h"

namespace gpu
{
	__device__ const float_t a1 = -7.91001919000e+00;
	__device__ const float_t s1 = 8.79671439570e-02;
	__device__ const float_t mu0 = global::mu0;
	__device__ const float_t pi = global::pi;

	__global__ void test_device_kernel(device_array* a, device_array* b, device_array* c)
	{
		int i = threadIdx.x;
		int n = a->size();

		auto da = a->get();
		auto db = b->get();
		auto dc = c->get();

		if (i >= n)
		{
			return;
		}
		// printf("%f = %f * %f\n", dc[i], da[i], db[i]);
		dc[i] = da[i] * db[i];
	}

	__global__ void test_sum_kernel(device_array* a)
	{
		int idx = threadIdx.x;
		int num = blockDim.x;

		auto res = a->get();

		int sum_num = num;
		//reduction
		while (sum_num > 1)
		{
			int id = idx;
			int next_sum_num = sum_num / 2;

			if ((sum_num & 0x1))
			{
				++id;
				++next_sum_num;
			}
			const int offset = sum_num / 2;
			if (idx < offset)
			{
				res[id] += res[id + offset];
			}
			sum_num = next_sum_num;
			__syncthreads();
		}
	}

	__device__ thrust::complex<float_t> return_dHz_w(float_t a, float_t i0, float_t h,
	                                                 device_array* hankel,
	                                                 device_array* resistivity,
	                                                 device_array* height,
	                                                 thrust::complex<float_t> w)
	{
		using complex = thrust::complex<float_t>;

		const float_t* hankel_ptr = hankel->get();
		const float_t* res_ptr = resistivity->get();
		const float_t* height_ptr = height->get();

		complex* ret = new complex(0, 0);

		const int res_size = resistivity->size();
		const int hankel_size = hankel->size();

		for (int k = 0; k < hankel_size; ++k)
		{
			const complex i(0, 1);
			const float_t lmd = 1 / a * pow(10.0, a1 + (k * s1));
			const float_t lmd_2 = pow(lmd, 2);

			const complex wi = i * w * mu0;
			const complex u1 = sqrt(lmd_2 - wi / res_ptr[0]);

			complex r0 = 1;

			for (int cc = res_size - 2; cc >= 0; --cc)
			{
				const complex ui = sqrt(lmd_2 - wi / res_ptr[cc]);
				const complex uii = sqrt(lmd_2 * lmd - wi / res_ptr[cc + 1]);

				const complex ss = ui / uii * r0;
				const complex ex1 = exp(-2 * ui * height_ptr[cc]);
				const complex ctan1 = (1 + ex1) / (1 - ex1);

				r0 = (1 + ctan1 * ss) / (ctan1 + ss);
			}
			const complex f1 = 1 + (lmd - u1 / r0) / (lmd + u1 / r0) * exp(-2 * lmd * h);

			*ret += f1 * lmd * hankel_ptr[k];
		}
		*ret = *ret * i0 / 2;

		auto ret_ = *ret;
		delete ret;
		return ret_;
	}


	__global__ void forward_kernel(float_t a, float_t i0, float_t h,
	                               device_array* cosine,
	                               device_array* hankel,
	                               device_array* resistivity,
	                               device_array* height,
	                               device_array* time,
	                               device_array* b)
	{
		const int time_idx = blockIdx.x;
		const int cosine_num = blockDim.x;
		const int cosine_idx = threadIdx.x;

		__shared__ float_t res[256];
		__shared__ float_t t;
		__shared__ float_t* cosine_ptr;

		if (cosine_idx == 0)
		{
			t = time->get()[time_idx];
			cosine_ptr = cosine->get();
		}

		__syncthreads();

		float_t w = 1 / t * exp((-150 + cosine_idx + 1) * std::log(10.0) / 20);
		thrust::complex<float_t> hz_w = return_dHz_w(a, i0, h, hankel, resistivity, height, w);

		res[cosine_idx] = hz_w.imag() / w * cosine_ptr[cosine_idx];

		__syncthreads();
		int sum_num = cosine_num;

		//reduction
		while (sum_num > 1)
		{
			int idx = cosine_idx;
			int next_sum_num = sum_num / 2;

			if ((sum_num & 0x1))
			{
				++idx;
				++next_sum_num;
			}
			const int offset = sum_num / 2;
			if (cosine_idx < offset)
			{
				res[idx] += res[idx + offset];
			}
			sum_num = next_sum_num;
			__syncthreads();
		}

		if (cosine_idx == 0)
		{
			b->get()[time_idx] = sqrt(2 / pi) / t * res[0];
		}
	}


	__global__ void calc_response_kernel(float_t a, float_t i0, float_t h,
	                                     device_array* b,
	                                     device_array* time,
	                                     device_array* response_late_m,
	                                     device_array* response_late_e)
	{
		const int time_idx = threadIdx.x;

		const auto b_ptr = b->get();
		const auto t = time->get()[time_idx];

		float_t* late_m_ptr = response_late_m->get();
		float_t* late_e_ptr = response_late_e->get();

		late_m_ptr[time_idx] =
			mu0 * pow(pi * i0 * pow(a, 2) / 30 / abs(b_ptr[time_idx]), 2.0 / 3) / pi / t;

		if (time_idx >= blockDim.x - 1)
		{
			return;
		}
		const auto t1 = time->get()[time_idx + 1];
		const auto t2 = (t + t1) / 2;
		const auto bt = (b_ptr[time_idx + 1] - b_ptr[time_idx]) / (t1 - t);

		late_e_ptr[time_idx] =
			mu0 * pow(2 * pi * i0 * pow(a, 2) / 5 / t2 / abs(bt), 2.0 / 3) / 4 / pi / t2;
	}

	void init_cuda_device()
	{
		int device_count;
		auto err = hipGetDeviceCount(&device_count);
		CHECK;
	}

	void test_cuda_device()
	{
		global::scoped_timer timer("test_cuda");

		global::log("test_cuda_device", "test start");

		device_array da;
		device_array db;
		device_array dc;

		vector a = {1, 2, 3, 4, 5};
		vector b = {1, 2, 3, 4, 5};
		vector c;

		const auto size = a.size();

		da.load_data(a);
		db.load_data(b);

		dc.allocate(size);

		global::log("test_cuda_device", "calculate start");

		test_device_kernel << <1, 32 >> >(da.get_device_ptr(), db.get_device_ptr(), dc.get_device_ptr());
		auto err = hipDeviceSynchronize();
		CHECK;

		dc.save_data(c);

		global::log("test_cuda_device", "calculate end");

		for (auto i = 0; i < size; ++i)
		{
			if (a[i] * b[i] != c[i])
			{
				throw std::runtime_error("test cuda device failed");
			}
		}

		vector s(250);
		for (int i = 0; i < s.size(); ++i)
		{
			s[i] = i;
		}
		device_array s_d(s);

		auto res = (s[0] + s[s.size() - 1]) * s.size() / 2;

		LOG("sum test start");
		test_sum_kernel << <1, s.size() >> >(s_d.get_device_ptr());
		err = hipDeviceSynchronize();
		CHECK;

		s_d.save_data(s);

		LOG("sum test end");

		if (s[0] != res)
		{
			throw std::runtime_error("test cuda device failed");
		}

		global::log("test_cuda_device", "test end");
	}

	void forward(float_t a, float_t i0, float_t h,
	             const vector& cosine, const vector& hankel,
	             const vector& resistivity, const vector& height,
	             const vector& time,
	             vector& response_late_m, vector& response_late_e)
	{
		device_array cosine_d(cosine);
		device_array hankel_d(hankel);

		device_array res_d(resistivity);
		device_array height_d(height);
		device_array time_d(time);

		device_array b(time.size());
		device_array late_m_d(time.size());
		device_array late_e_d(time.size());

		forward_kernel << <time_d.size(), cosine_d.size()>> >(
			a, i0, h,
			cosine_d.get_device_ptr(), hankel_d.get_device_ptr(),
			res_d.get_device_ptr(), height_d.get_device_ptr(),
			time_d.get_device_ptr(), b.get_device_ptr());
		auto err = hipDeviceSynchronize();
		CHECK;

#if defined(_DEBUG)
		vector test_b;
		b.save_data(test_b);
#endif

		calc_response_kernel << <1, time_d.size() >> >(
			a, i0, h,
			b.get_device_ptr(),
			time_d.get_device_ptr(),
			late_m_d.get_device_ptr(),
			late_e_d.get_device_ptr());
		err = hipDeviceSynchronize();
		CHECK;

		late_m_d.save_data(response_late_m);
		late_e_d.save_data(response_late_e);
	}
}
