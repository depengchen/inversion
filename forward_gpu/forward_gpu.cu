#include "hip/hip_runtime.h"
﻿#include <cmath>
#include <memory>
#include <complex>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include "forward_gpu.h"
#include "device_ptr.h"
#include "device_array.h"
#include "cuda_helper.h"

namespace gpu
{
	__device__ const float_t a1 = -7.91001919000e+00;
	__device__ const float_t s1 = 8.79671439570e-02;
	__device__ const float_t mu0 = global::mu0;
	__device__ const float_t pi = global::pi;

	__global__ void test_device_kernel(float_t* a, float_t* b, float_t* c, int num)
	{
		int i = threadIdx.x;
		if (i >= num)
		{
			return;
		}
		c[i] = a[i] * b[i];
	}

	__device__ thrust::complex<float_t> return_dHz_w(float_t a, float_t i0, float_t h,
	                                                 device_array* hankel,
	                                                 device_array* resistivity,
	                                                 device_array* height,
	                                                 thrust::complex<float_t> w)
	{
		using complex = thrust::complex<float_t>;

		complex ret(0, 0);

		const float_t* hankel_ptr = hankel->get();
		const float_t* res_ptr = resistivity->get();
		const float_t* height_ptr = height->get();

		const int res_size = resistivity->size();
		const int hankel_size = hankel->size();

		for (int k = 0; k < hankel_size; ++k)
		{
			const complex i(0, 1);
			const float_t lmd = 1 / a * powf(10, a1 + (k * s1));

			const complex u1 = sqrt(powf(lmd, 2) - i * w * mu0 / res_ptr[0]);

			complex r0 = 1;
			for (int cc = res_size - 2; cc > 00; --cc)
			{
				const float_t lmd_2 = pow(lmd, 2);
				const complex wi = i * w * mu0;

				const complex ui = sqrt(lmd_2 - wi / res_ptr[cc]);
				const complex uii = sqrt(lmd_2 - wi / res_ptr[cc + 1]);

				const complex ss = ui / uii * r0;
				const complex ex1 = exp(-2 * ui * height_ptr[cc]);
				const complex ctan1 = (1 + ex1) / (1 - ex1);

				r0 = (1 + ctan1 * ss) / (ctan1 + ss);
			}
			const complex f1 = 1 + (lmd - u1 / r0) / (lmd + u1 / r0) * exp(-2 * lmd * h);

			ret += f1 * lmd * hankel_ptr[k];
		}
		ret = ret * i0 / 2;
		return ret;
	}

	/**
	 * \brief 计算正演kernel函数
	 * \param a 回线半径(m)
	 * \param i0 发射电流(A)
	 * \param h 发射、接收回线高度(m)
	 * \param cosine 余弦变换系数
	 * \param hankel 汉克尔变换系数
	 * \param resistivity 地层电阻率
	 * \param height 地层厚度
	 * \param time 时间
	 * \param response_late_m 晚期磁场响应
	 * \param response_late_e 晚期电场响应
	 */
	__global__ void forward_kernel(float_t a, float_t i0, float_t h,
	                               device_array* cosine,
	                               device_array* hankel,
	                               device_array* resistivity,
	                               device_array* height,
	                               device_array* time,
	                               device_array* response_late_m,
	                               device_array* response_late_e)
	{
		const int time_idx = blockIdx.x;
		const int cosine_num = blockDim.x;
		const int cosine_idx = threadIdx.x;

		extern __shared__ float_t res_complex[];
		__shared__ float_t t;
		__shared__ float_t* cosine_ptr;

		//每个block中的第一个线程为计算准备数据
		if (cosine_idx == 0)
		{
			t = time->get()[time_idx];
			cosine_ptr = cosine->get();
		}

		__syncthreads();

		float_t w = 1 / t * exp((-150 + cosine_idx - 1) * std::log(10.0) / 20);
		thrust::complex<float_t> hz_w = return_dHz_w(a, i0, h, hankel, resistivity, height, w);

		res_complex[cosine_idx] = hz_w.imag() / w * cosine_ptr[cosine_idx];

		//二分求和
		for (int offset = cosine_num / 2; offset > 0; offset >>= 1)
		{
			if (cosine_idx < offset)
			{
				res_complex[cosine_idx] += res_complex[cosine_idx + offset];
			}
			__syncthreads();
		}

		//每个block中的第一个线程做收尾计算
		if (cosine_idx == 0)
		{
			const float_t dHz = sqrt(2 / pi) / t * res_complex[0];

			if (response_late_m)
				response_late_m->get()[time_idx] =
					mu0 * pow(pi * i0 * pow(a, 2) / 30 / abs(dHz), 2.0 / 3) / pi / t;
		}
	}

	void init_cuda_device()
	{
		int device_count;
		auto err = hipGetDeviceCount(&device_count);
		CHECK;
		global::log("forward", "init_cuda_device complete");
	}

	void test_cuda_device()
	{
		global::scoped_timer("test_cuda");

		global::log("test_cuda_device", "test start");

		device_ptr<float_t> da;
		device_ptr<float_t> db;
		device_ptr<float_t> dc;

		float_t a[] = {1, 2, 3, 4, 5};
		float_t b[] = {1, 2, 3, 4, 5};

		auto size = sizeof(a) / sizeof(float_t);
		float_ptr c(new float_t[size]);

		da.allocate(size);
		db.allocate(size);
		dc.allocate(size);

		copy_to_device(a, da.get(), size);
		copy_to_device(b, db.get(), size);

		global::log("test_cuda_device", "calculate start");

		test_device_kernel << <1, 32 >> >(da.get(), db.get(), dc.get(), size);
		auto err = hipDeviceSynchronize();
		CHECK;

		copy_to_host(dc.get(), c.get(), size);

		global::log("test_cuda_device", "calculate end");

		for (auto i = 0; i < size; ++i)
		{
			if (a[i] * b[i] != c[i])
			{
				throw std::runtime_error("测试cuda设备失败，计算错误");
			}
		}
		global::log("test_cuda_device", "test end");
	}

	void forward(float_t a, float_t i0, float_t h,
	             const vector& cosine, const vector& hankel,
	             const vector& resistivity, const vector& height,
	             const vector& time,
	             vector& response_late_m, vector& response_late_e)
	{
		device_array cosine_d(cosine);
		device_array hankel_d(hankel);

		device_array res_d(resistivity);
		device_array height_d(height);
		device_array time_d(time);

		device_array late_m_d(time.size());
		device_array late_e_d(time.size());

		forward_kernel << <time_d.size(), cosine_d.size(), sizeof(float_t) * cosine_d.size() >> >(
			a, i0, h,
			cosine_d.get_device_ptr(), hankel_d.get_device_ptr(),
			res_d.get_device_ptr(), height_d.get_device_ptr(),
			time_d.get_device_ptr(),
			late_m_d.get_device_ptr(), late_e_d.get_device_ptr());

		late_m_d.save_data(response_late_m);
		late_e_d.save_data(response_late_e);
	}
}
